#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <ctime>


#define N 65535


__global__ void add(int *a)
{
	/*int tid = blockIdx.x;
	if (tid < N)
		a[tid] = tid + 1;*/

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < N)
	{
		a[tid] = tid + 1;
		tid += blockDim.x * gridDim.x;
	}
}


int main(void)
{
	int a[N];
	int *dev_a;

	int threadsPerBlock = 1024;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	hipEvent_t start, stop;
	float gpuTime = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipMalloc((void**)&dev_a, N * sizeof(int));

	add<<<blocksPerGrid, threadsPerBlock >>>(dev_a);

	hipMemcpy(a, dev_a, N * sizeof(int), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);


	for (int i = 0; i < N; ++i)
		printf("%d\t", a[i]);

	printf("\n===================   GPU    ===================\n");
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	printf("DEVICE GPU compute time: %.2f milliseconds\n\n", gpuTime);

	hipFree(dev_a);


	// CPU
	int start2, time2;
	start2 = clock();

	int a2[N];

	for (int i = 0; i < N; ++i)
		a2[i] = i + 1;
		
	time2 = clock() - start2;

	//for (int i = 0; i < N; ++i)
		//printf("%d\t", a2[i]);

	double time_CPU = time2 / 2.0;

	printf("\n===================   CPU    ===================\n");
	printf("CPU compute time: %f milliseconds\n\n", time_CPU);

	return 0;
}