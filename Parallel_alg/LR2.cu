#include "hip/hip_runtime.h"
#include <stdio.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

#define CSC(call) do {              \
    hipError_t err = call;           \
    if (err != hipSuccess) {       \
        fprintf(stderr, "CUDA error in file %s in line %d: %s.\n", __FILE__, __LINE__, hipGetErrorString(err));    \
        exit(0);                    \
    }                               \
} while(0)

__global__ void kernel_radix(int *arr, int *radix, int n, int k) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x;
    while(idx < n) {
        radix[idx] = (arr[idx] >> k) & 1;
        idx += offset;
    }
}

__global__ void kernel_perm(int *arr, int *radix, int *out, int n, int k) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x;
    int sn = radix[n - 1] + ((arr[n - 1] >> k) & 1);
    while(idx < n) {
        if ((arr[idx] >> k) & 1)
            out[radix[idx] + (n - sn)] = arr[idx];
        else
            out[idx - radix[idx]] = arr[idx];
        idx += offset;
    }
}

void radix_sort(int *dev_arr, int n) {
    int *dev_out, *dev_radix, *temp;
    CSC(hipMalloc(&dev_out, sizeof(int) * n));
    CSC(hipMalloc(&dev_radix, sizeof(int) * n));
    int k;
    for(k = 0; k < 32; k++) {
        kernel_radix<<<256, 256>>>(dev_arr, dev_radix, n, k);
        thrust::exclusive_scan(thrust::device, dev_radix, dev_radix + n, dev_radix);
        kernel_perm<<<256, 256>>>(dev_arr, dev_radix, dev_out, n, k);
        temp = dev_arr;
        dev_arr = dev_out;
        dev_out = temp;
    }
}

int main() {
    int i, n = 10000;
    int *dev_arr, *arr = (int *)malloc(sizeof(int) * n);
    for(i = 0; i < n; i++)
        arr[i] = (i * i) % 10000;
    CSC(hipMalloc(&dev_arr, sizeof(int) * n));
    CSC(hipMemcpy(dev_arr, arr, sizeof(int) * n, hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    float time;
    CSC(hipEventCreate(&start));
    CSC(hipEventCreate(&stop));
    CSC(hipEventRecord(start, 0));

    radix_sort(dev_arr, n);

    CSC(hipGetLastError());
    CSC(hipEventRecord(stop, 0));
    CSC(hipEventSynchronize(stop));
    CSC(hipEventElapsedTime(&time, start, stop));
    CSC(hipEventDestroy(start));
    CSC(hipEventDestroy(stop));

    printf("time = %f\n", time);

    CSC(hipMemcpy(arr, dev_arr, sizeof(int) * n, hipMemcpyDeviceToHost));
    for(i = n - 100; i < n; i++)
        printf("%d ", arr[i]);
    printf("\n");

    CSC(hipFree(dev_arr));
    free(arr);
    return 0;
}